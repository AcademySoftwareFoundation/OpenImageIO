#include "hip/hip_runtime.h"
/*
  Copyright 2018 Larry Gritz and the other authors and contributors.
  All Rights Reserved.

  Redistribution and use in source and binary forms, with or without
  modification, are permitted provided that the following conditions are
  met:
  * Redistributions of source code must retain the above copyright
    notice, this list of conditions and the following disclaimer.
  * Redistributions in binary form must reproduce the above copyright
    notice, this list of conditions and the following disclaimer in the
    documentation and/or other materials provided with the distribution.
  * Neither the name of the software's owners nor the names of its
    contributors may be used to endorse or promote products derived from
    this software without specific prior written permission.
  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
  "AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
  LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
  A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
  OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
  SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
  LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
  DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
  THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
  OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

  (This is the Modified BSD License)
*/

#include <OpenImageIO/imagebuf.h>
#include <OpenImageIO/timer.h>
#include "imagebufalgo_cuda.h"


OIIO_NAMESPACE_BEGIN
namespace pvt {


__global__
void add_cuda (float *R, const float *A, const float *B, ROI roi)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int n = int(roi.npixels());
    int nc = roi.nchannels();
    for (int p = index; p < n; p += stride) {
        int i = p*nc;
        for (int c = roi.chbegin; c < roi.chend; ++c)
            R[i+c] = A[i+c] + B[i+c];
    }
}



bool
add_impl_cuda (ImageBuf &R, const ImageBuf &A, const ImageBuf &B,
               ROI roi)
{
    Timer timer;
    int blockSize = 1024;
    int numBlocks = (int(roi.npixels()) + blockSize - 1) / blockSize;
    add_cuda<<<numBlocks, blockSize>>>((float *)R.localpixels(),
                                       (const float *)A.localpixels(),
                                       (const float *)B.localpixels(), roi);
    hipDeviceSynchronize();
    OIIO::debug ("Running cuda ImageBufAlgo::add, %d blocks of %d: %gms\n",
                 numBlocks, blockSize, timer()*1000.0f);
    return true;
}




__global__
void sub_cuda (float *R, const float *A, const float *B, ROI roi)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    int n = int(roi.npixels());
    int nc = roi.nchannels();
    for (int p = index; p < n; p += stride) {
        int i = p*nc;
        for (int c = roi.chbegin; c < roi.chend; ++c)
            R[i+c] = A[i+c] - B[i+c];
    }
}



bool
sub_impl_cuda (ImageBuf &R, const ImageBuf &A, const ImageBuf &B,
               ROI roi)
{
    Timer timer;
    int blockSize = 1024;
    int numBlocks = (int(roi.npixels()) + blockSize - 1) / blockSize;
    sub_cuda<<<numBlocks, blockSize>>>((float *)R.localpixels(),
                                       (const float *)A.localpixels(),
                                       (const float *)B.localpixels(), roi);
    hipDeviceSynchronize();
    OIIO::debug ("Running cuda ImageBufAlgo::sub, %d blocks of %d: %gms\n",
                 numBlocks, blockSize, timer()*1000.0f);
    return true;
}


}  // end namespace pvt
OIIO_NAMESPACE_END

